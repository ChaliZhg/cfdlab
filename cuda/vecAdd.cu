
#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void VecAdd(float* A, float* B, float *C)
{
   int idx = blockIdx.x*blockDim.x + threadIdx.x;
   C[idx] = A[idx] + B[idx];
}

int main()
{
   int N=16*16*16;
   float *A=0, *B=0, *C=0;
   float *Ad=0, *Bd=0, *Cd=0;

   A = (float*)malloc(N*sizeof(float));
   B = (float*)malloc(N*sizeof(float));
   C = (float*)malloc(N*sizeof(float));
   if(0==A||0==B||0==C){
      printf("Could not allocate host memory\n");
      return 1;
   }

   for(int i=0; i<N; i++)
   {
      A[i] = i;
      B[i] = i;
   }

   hipMalloc( (void**)&Ad, N*sizeof(float) );
   hipMalloc( (void**)&Bd, N*sizeof(float) );
   hipMalloc( (void**)&Cd, N*sizeof(float) );

   if(0==Ad||0==Bd||0==Cd){
      printf("Could not allocate device memory\n");
      return 2;
   }

   hipMemcpy(Ad, A, N*sizeof(float), hipMemcpyHostToDevice);
   hipMemcpy(Bd, B, N*sizeof(float), hipMemcpyHostToDevice);

   dim3 grid, block;
   block.x = 1;
   grid.x  = N/block.x;

   VecAdd<<<grid, block>>>(Ad, Bd, Cd);

   hipMemcpy(C, Cd, N*sizeof(float), hipMemcpyDeviceToHost);

   for(int i=0; i<N; i++)
      printf("%d %e %e %e\n", i, A[i], B[i], C[i]);

   free(A); free(B); free(C);
   hipFree(Ad); hipFree(Bd); hipFree(Cd);

   return 0;
}
