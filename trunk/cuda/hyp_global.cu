/* Linear convection equation with periodic BC
 * solved using MUSCL scheme
 * CUDA implementation of hyp.c using only global memory
*/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#define minmod(a,b)   ( (fabs(a) < fabs(b)) ? (a) : (b) )


//function prototypes
void initCond(int, float, float*);
__global__ void fluxFun(float*, float*);
__global__ void update(int, float, float, float*, float*, float*);
__global__ void periodic(float*, int);

int main(){
   float *u;
   float *uold_d, *u_d, *fl_d;
   int np = 101, ns;
   float dx = 1.0/(np-1);
   float dt, cfl;
   int niter, maxiter;
   int nirk, rkmax=3;
   int i;
   FILE *fpt;
   dim3 grid, block;

   ns   = np + 2 + 2;

   u    = (float*)malloc(ns*sizeof(float));

   hipMalloc((void**)&uold_d, (ns)*sizeof(float));
   hipMalloc((void**)&u_d,    (ns)*sizeof(float));
   hipMalloc((void**)&fl_d,   (np+1)*sizeof(float));

   cfl = 0.9;
   dt  = cfl*dx;
   maxiter = 1.0/dt + 1;

   //set initial conditions
   initCond(np, dx, u);

   fpt = fopen("init.dat", "w");
   for(i=0; i<np; i++) fprintf(fpt, "%e %e\n", dx*i, u[i+2]);
   fclose(fpt);

   hipMemcpy(uold_d, u, (ns)*sizeof(float), 
              hipMemcpyHostToDevice);
   hipMemcpy(u_d, u, (ns)*sizeof(float), 
              hipMemcpyHostToDevice);

   //time-step loop
   for(niter=0; niter<maxiter; niter++){

      //RK stages
      for(nirk=0; nirk<rkmax; nirk++){

         //flux computation
         block.x = 3;
         grid.x  = (np+1)/block.x;
         fluxFun<<<grid,block>>>(u_d, fl_d);

         //update conserved variable
         block.x = 1;
         grid.x  = (np)/block.x;
         update<<<grid,block>>>(nirk, dt, dx, uold_d, fl_d, u_d);

         //set periodicity
         block.x = 1;
         grid.x  = (ns)/block.x;
         periodic<<<grid,block>>>(u_d, np);
      }
      hipMemcpy(uold_d, u_d, (ns)*sizeof(float), 
                 hipMemcpyDeviceToDevice);

   }
   hipMemcpy(u, u_d, (ns)*sizeof(float), 
              hipMemcpyDeviceToHost);

   fpt = fopen("final.dat", "w");
   for(i=0; i<np; i++) fprintf(fpt, "%e %e\n", dx*i, u[i+2]);
   fclose(fpt);

   free(u);

   hipFree(uold_d);
   hipFree(u_d);
   hipFree(fl_d);

}

//set initial condition
void initCond(int np, float dx, float *u){
   int i;
   float x;

   for(i=0; i<np; i++){
      x = dx*i;
      u[i+2] = sin(2.0*M_PI*x);
   }
   u[0]    = u[np];
   u[1]    = u[np+1];
   u[np+2] = u[2];
   u[np+3] = u[3];

}

//flux function
__global__ void fluxFun(float *u, float *fl){
   float uj, ujp1, ujm1, ujp2;
   float ul, ur;

   int idx = blockIdx.x*blockDim.x + threadIdx.x;

   ujm1 = *(u+idx);
   uj   = *(u+idx+1);
   ujp1 = *(u+idx+2);
   ujp2 = *(u+idx+3);

   ul = uj   + 0.5*minmod( (uj-ujm1), (ujp1-uj) );
   ur = ujp1 - 0.5*minmod( (ujp1-uj), (ujp2-ujp1) );

   fl[idx] = ul;

}

//perform one stage of RK
__global__ void update(int nirk, float dt, float dx, float *uold, float *fl, 
                       float *u){
   int idx = blockIdx.x*blockDim.x + threadIdx.x;
   float res;
   float airk[3] = {0.0, 3.0/4.0, 1.0/3.0};

   res = fl[idx+1] - fl[idx];
   u[idx+2] = airk[nirk]*uold[idx+2] + 
          (1.0-airk[nirk])*(u[idx+2] - (dt/dx)*res);
}

//set periodic BC
__global__ void periodic(float *u, int np){
   int idx = blockIdx.x*blockDim.x + threadIdx.x;

   if(idx==0)
      u[idx] = u[np];
   else if(idx==1)
      u[idx] = u[np+1];
   else if(idx==np+2)
      u[idx] = u[2];
   else if(idx==np+3)
      u[idx] = u[3];
   else
      u[idx] = u[idx];

}
